#include "hip/hip_runtime.h"
// %%writefile parallel.cu

#include <stdio.h>
#include <stdlib.h>
#include "opencv2/opencv.hpp"
#include <chrono>

using namespace cv;
using namespace std;

__device__ int b(int a, int b, int c) {
    int positive_count = 0;

    if (a > 0) {
        positive_count++;
    }
    if (b > 0) {
        positive_count++;
    }
    if (c > 0) {
        positive_count++;
    }

    return (positive_count >= 2) ? 1 : 0;
}

__global__ void ltpcKernel(uchar* input, uchar* output, int rows, int cols) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x > 0 && x < cols && y > 0 && y < rows) {
        int C = input[y * cols + x];
        int P0 = input[y * cols + x + 1];
        int P1 = input[(y + 1) * cols + x + 1];
        int P2 = input[(y + 1) * cols + x];
        int P3 = input[(y + 1) * cols + x - 1];
        int P4 = input[y * cols + x - 1];
        int P5 = input[(y - 1) * cols + x - 1];
        int P6 = input[(y - 1) * cols + x];
        int P7 = input[(y - 1) * cols + x + 1];

        int CP[8];
        CP[0] = b(P7 - C, P0 - C, P1 - C);
        CP[1] = b(P1 - C, P2 - C, P3 - C);
        CP[2] = b(P3 - C, P4 - C, P5 - C);
        CP[3] = b(P5 - C, P6 - C, P7 - C);
        CP[4] = b(P6 - P0, C - P0, P2 - P0);
        CP[5] = b(P4 - P2, C - P2, P0 - P2);
        CP[6] = b(P2 - P4, C - P4, P6 - P4);
        CP[7] = b(P0 - P6, C - P6, P4 - P6);

        int decimal_value = 0;
        for (int i = 7; i >= 0; i--) {
            decimal_value = decimal_value * 2 + CP[i];
        }

        output[y * cols + x] = static_cast<uchar>(decimal_value);
    }
}

int main() {
    auto start_time = std::chrono::high_resolution_clock::now();
    Mat image = imread("small_ip.png", IMREAD_GRAYSCALE);
    if (image.empty()) {
        printf("Image not found or could not be opened.\n");
        return 1;
    }
    auto image_load_time = std::chrono::high_resolution_clock::now();
    auto image_loading_duration = std::chrono::duration_cast<std::chrono::milliseconds>(image_load_time - start_time);

    int rows = image.rows;
    int cols = image.cols;

    Mat result_image(rows, cols, CV_8UC1);

    uchar* d_input, *d_output;
    hipMalloc((void**)&d_input, rows * cols * sizeof(uchar));
    hipMalloc((void**)&d_output, rows * cols * sizeof(uchar));

    hipMemcpy(d_input, image.data, rows * cols * sizeof(uchar), hipMemcpyHostToDevice);

    dim3 blockDim(16, 16);
    dim3 gridDim((cols + blockDim.x - 1) / blockDim.x, (rows + blockDim.y - 1) / blockDim.y);

    ltpcKernel<<<gridDim, blockDim>>>(d_input, d_output, rows, cols);

    hipMemcpy(result_image.data, d_output, rows * cols * sizeof(uchar), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);

    auto end_time = std::chrono::high_resolution_clock::now();

    auto total_duration = std::chrono::duration_cast<std::chrono::milliseconds>(end_time - start_time);

    printf("Image loading time: %ld ms\n", image_loading_duration.count());
    printf("LTCP descriptor calculation time: %ld ms\n", total_duration.count() - image_loading_duration.count());
    printf("Total running time: %ld ms\n", total_duration.count());

    imwrite("result_image.jpg", result_image);
    cout<<result_image;

    return 0;
}
